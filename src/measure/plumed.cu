
#include <hip/hip_runtime.h>
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Interface to the PLUMED plugin: https://www.plumed.org
------------------------------------------------------------------------------*/

#ifdef USE_PLUMED

#include "plumed.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/gpu_vector.cuh"
#include "utilities/read_file.cuh"
#include <cstring>

#define E_C 1.602176634E-19 // Elementary charge
#define N_A 6.0221367E23    // Avogadro constant

const double ENERGY_UNIT_CONVERSION = N_A * E_C / 1000; // from eV to kJ/mol

static __global__ void gpu_sum(const int N, const double* g_data, double* g_data_sum)
{
  int number_of_rounds = (N - 1) / 1024 + 1;
  __shared__ double s_data[1024];
  s_data[threadIdx.x] = 0.0;
  for (int round = 0; round < number_of_rounds; ++round) {
    int n = threadIdx.x + round * 1024;
    if (n < N) {
      s_data[threadIdx.x] += g_data[n + blockIdx.x * N];
    }
  }
  __syncthreads();
  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset) {
      s_data[threadIdx.x] += s_data[threadIdx.x + offset];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    g_data_sum[blockIdx.x] = s_data[0];
  }
}

static void __global__ gpu_scale_virial(
  const int N,
  const double* factors,
  double* g_sxx,
  double* g_syy,
  double* g_szz,
  double* g_sxy,
  double* g_sxz,
  double* g_syz)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    g_sxx[i] *= factors[0];
    g_syy[i] *= factors[4];
    g_szz[i] *= factors[8];
    g_sxy[i] *= factors[1];
    g_sxz[i] *= factors[2];
    g_syz[i] *= factors[5];
  }
}

void PLUMED::preprocess(
  const int number_of_steps,
  const double time_step,
  Integrate& integrate,
  std::vector<Group>& group,
  Atom& atom,
  Box& box,
  Force& force)
{
  n_atom = atom.number_of_atoms;
  gpu_v_vector.resize(6);
  gpu_v_factor.resize(9);
  cpu_b_vector = std::vector<double>(9);
  cpu_v_vector = std::vector<double>(9);
  cpu_v_factor = std::vector<double>(9);
  cpu_m_vector = std::vector<double>(3 * n_atom);
  cpu_f_vector = std::vector<double>(3 * n_atom);
  cpu_q_vector = std::vector<double>(3 * n_atom);
  memcpy(cpu_m_vector.data(), atom.cpu_mass.data(), n_atom * sizeof(double));

  init(time_step, integrate.temperature);
}

PLUMED::PLUMED(const char** param, int num_param)
{
  parse(param, num_param);
  property_name = "plumed";
}

void PLUMED::parse(const char** param, int num_param)
{
  use_plumed = 1;
  memset(input_file, 0, 80);
  if (!plumed_installed()) {
    PRINT_INPUT_ERROR("PLUMED not installed!\n");
  }
  if (num_param != 4) {
    PRINT_INPUT_ERROR("plumed should have 3 parameters.");
  }
  sprintf(input_file, "%s", param[1]);
  sprintf(output_file, "%s.out", param[1]);
  if (!is_valid_int(param[2], &interval)) {
    PRINT_INPUT_ERROR("plumed invoke interval should be an integer.");
  }
  if (interval <= 0) {
    PRINT_INPUT_ERROR("plumed invoke interval should > 0.");
  }
  if (!is_valid_int(param[3], &restart)) {
    PRINT_INPUT_ERROR("plumed restart parameter should be 0 or 1.");
  }
  printf("Use PLUMED for this run.\n");
  printf("    input  file: '%s'.\n", input_file);
  printf("    output file: '%s'.\n", output_file);
  printf("    invoke freq: every %d steps.\n", interval);
  if (restart) {
    printf("    will restart calculations from old files.\n");
  }
}

void PLUMED::init(const double ts, const double T)
{
  step = 0;
  time_step = ts;

  const char engine_name[7] = "GPUMD\0";                // my name
  const double KbT = K_B * T;                           // eV
  const double time_unit = TIME_UNIT_CONVERSION / 1000; // natural -> ps
  const double mass_unit = 1.0;                         // amu. -> amu.
  const double energy_unit = ENERGY_UNIT_CONVERSION;    // ev -> kJ/mol
  const double length_unit = 0.1;                       // Ang -> nm
  const double charge_unit = 1.0;                       // e -> e

  plumed_main = plumed_create();
  plumed_cmd(plumed_main, "setKbT", &KbT);
  plumed_cmd(plumed_main, "setMDEngine", engine_name);
  plumed_cmd(plumed_main, "setMDTimeUnits", &time_unit);
  plumed_cmd(plumed_main, "setMDMassUnits", &mass_unit);
  plumed_cmd(plumed_main, "setMDEnergyUnits", &energy_unit);
  plumed_cmd(plumed_main, "setMDLengthUnits", &length_unit);
  plumed_cmd(plumed_main, "setMDChargeUnits", &charge_unit);
  plumed_cmd(plumed_main, "setPlumedDat", input_file);
  plumed_cmd(plumed_main, "setLogFile", output_file);
  plumed_cmd(plumed_main, "setTimestep", &time_step);
  plumed_cmd(plumed_main, "setRestart", &restart);
  plumed_cmd(plumed_main, "setNatoms", &n_atom);
  plumed_cmd(plumed_main, "init", NULL);
}

void PLUMED::process(
  const int number_of_steps,
  int step_input,
  const int fixed_group,
  const int move_group,
  const double global_time,
  const double temperature,
  Integrate& integrate,
  Box& box,
  std::vector<Group>& group,
  GPU_Vector<double>& thermo,
  Atom& atom,
  Force& force)
{
  if (step_input % interval != 0) {
    return;
  }

  std::vector<double> tmp(6);
  step += interval;

  atom.force_per_atom.copy_to_host(cpu_f_vector.data());
  atom.position_per_atom.copy_to_host(cpu_q_vector.data());

  cpu_b_vector[0] = box.cpu_h[0];
  cpu_b_vector[1] = box.cpu_h[3];
  cpu_b_vector[2] = box.cpu_h[6];
  cpu_b_vector[3] = box.cpu_h[1];
  cpu_b_vector[4] = box.cpu_h[4];
  cpu_b_vector[5] = box.cpu_h[7];
  cpu_b_vector[6] = box.cpu_h[2];
  cpu_b_vector[7] = box.cpu_h[5];
  cpu_b_vector[8] = box.cpu_h[8];

  gpu_sum<<<6, 1024>>>(n_atom, atom.virial_per_atom.data(), gpu_v_vector.data());
  GPU_CHECK_KERNEL
  gpu_v_vector.copy_to_host(tmp.data());
  fill(cpu_v_vector.begin(), cpu_v_vector.end(), 0.0);

  plumed_cmd(plumed_main, "setStep", &step);
  plumed_cmd(plumed_main, "setMasses", cpu_m_vector.data());
  plumed_cmd(plumed_main, "setBox", cpu_b_vector.data());
  plumed_cmd(plumed_main, "setVirial", cpu_v_vector.data());
  plumed_cmd(plumed_main, "setForcesX", &(cpu_f_vector.data()[0 * n_atom]));
  plumed_cmd(plumed_main, "setForcesY", &(cpu_f_vector.data()[1 * n_atom]));
  plumed_cmd(plumed_main, "setForcesZ", &(cpu_f_vector.data()[2 * n_atom]));
  plumed_cmd(plumed_main, "setPositionsX", &(cpu_q_vector.data()[0 * n_atom]));
  plumed_cmd(plumed_main, "setPositionsY", &(cpu_q_vector.data()[1 * n_atom]));
  plumed_cmd(plumed_main, "setPositionsZ", &(cpu_q_vector.data()[2 * n_atom]));
  plumed_cmd(plumed_main, "prepareCalc", NULL);
  plumed_cmd(plumed_main, "performCalc", NULL);
  plumed_cmd(plumed_main, "getBias", &bias_energy);
  plumed_cmd(plumed_main, "setStopFlag", &stop_flag);

  atom.force_per_atom.copy_from_host(cpu_f_vector.data());

  cpu_v_factor[0] = (tmp[0] - cpu_v_vector[0]) / tmp[0];
  cpu_v_factor[1] = (tmp[3] - cpu_v_vector[1]) / tmp[3];
  cpu_v_factor[2] = (tmp[4] - cpu_v_vector[2]) / tmp[4];
  cpu_v_factor[3] = (tmp[3] - cpu_v_vector[3]) / tmp[3];
  cpu_v_factor[4] = (tmp[1] - cpu_v_vector[4]) / tmp[1];
  cpu_v_factor[5] = (tmp[5] - cpu_v_vector[5]) / tmp[5];
  cpu_v_factor[6] = (tmp[4] - cpu_v_vector[6]) / tmp[4];
  cpu_v_factor[7] = (tmp[5] - cpu_v_vector[7]) / tmp[5];
  cpu_v_factor[8] = (tmp[2] - cpu_v_vector[8]) / tmp[2];
  gpu_v_factor.copy_from_host(cpu_v_factor.data());
  gpu_scale_virial<<<(n_atom - 1) / 128 + 1, 128>>>(
    n_atom,
    gpu_v_factor.data(),
    atom.virial_per_atom.data() + n_atom * 0,
    atom.virial_per_atom.data() + n_atom * 1,
    atom.virial_per_atom.data() + n_atom * 2,
    atom.virial_per_atom.data() + n_atom * 3,
    atom.virial_per_atom.data() + n_atom * 4,
    atom.virial_per_atom.data() + n_atom * 5);
  GPU_CHECK_KERNEL
}

void PLUMED::postprocess(
  Atom& atom,
  Box& box,
  Integrate& integrate,
  const int number_of_steps,
  const double time_step,
  const double temperature)
{
  if (use_plumed == 1) {
    use_plumed = 0;
    plumed_finalize(plumed_main);
  }
}

#endif
